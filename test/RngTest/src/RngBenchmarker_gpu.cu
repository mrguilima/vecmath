#include "hip/hip_runtime.h"
#include "VecMath/Rng/MRG32k3a.h"
#include "VecMath/Rng/Philox.h"
#include "VecMath/Rng/Threefry.h"

namespace vecrng {
inline namespace cuda {

const int THREADS_PER_BLOCK = 256;

// do reduction on GPU: THREADS_PER_BLOCK should be 2^N
#define __reduction_on_gpu(Input, Index, Result)                               \
  {                                                                            \
    __syncthreads();                                                           \
    int i = blockDim.x / 2;                                                    \
    while (i != 0) {                                                           \
      if (Index < i)                                                           \
        Input[Index] += Input[Index + i];                                      \
      __syncthreads();                                                         \
      i /= 2;                                                                  \
    }                                                                          \
    if (sid == 0)                                                              \
      Result = Input[0];                                                       \
  }

__global__ void
KernelMRG32k3a(vecRng::MRG32k3a<vecRng::ScalarBackend>::State_t *devStates,
               double *result, int nsample) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = threadIdx.x;

  vecRng::MRG32k3a<vecRng::ScalarBackend> rng(0);

  __shared__ double sum[THREADS_PER_BLOCK];
  double tmp = 0;

  while (tid < nsample) {
    tmp += rng.Uniform<vecRng::ScalarBackend>(&devStates[sid]);
    tid += blockDim.x * gridDim.x;
  }
  sum[sid] = tmp;

  // do reduction on GPU
  __reduction_on_gpu(sum, sid, result[blockIdx.x]);
}

__global__ void
KernelThreefry(vecRng::Threefry<vecRng::ScalarBackend>::State_t *devStates,
               double *result, int nsample) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = threadIdx.x;

  vecRng::Threefry<vecRng::ScalarBackend> rng(0);

  __shared__ double sum[THREADS_PER_BLOCK];
  double tmp = 0;

  while (tid < nsample) {
    tmp += rng.Uniform<vecRng::ScalarBackend>(&devStates[sid]);
    tid += blockDim.x * gridDim.x;
  }
  sum[sid] = tmp;

  // do reduction on GPU
  __reduction_on_gpu(sum, sid, result[blockIdx.x]);
}

__global__ void
KernelPhilox(vecRng::Philox<vecRng::ScalarBackend>::State_t *devStates,
             double *result, int nsample) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = threadIdx.x;

  vecRng::Philox<vecRng::ScalarBackend> rng(0);

  __shared__ double sum[THREADS_PER_BLOCK];
  double tmp = 0;

  while (tid < nsample) {
    tmp += rng.Uniform<vecRng::ScalarBackend>(&devStates[sid]);
    tid += blockDim.x * gridDim.x;
  }
  sum[sid] = tmp;

  // do reduction on GPU
  __reduction_on_gpu(sum, sid, result[blockIdx.x]);
}

//-----------------------------------------------------------------------------
//  Curand MRG32k3a
//-----------------------------------------------------------------------------
__global__ void KernelCurandMRG32k3a(hiprandStateMRG32k3a *devStates,
                                     double *result, int nsample) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = threadIdx.x;

  hiprandStateMRG32k3a localState = devStates[sid];

  __shared__ double sum[THREADS_PER_BLOCK];
  double tmp = 0;

  while (tid < nsample) {
    tmp += hiprand_uniform_double(&localState);
    tid += blockDim.x * gridDim.x;
  }
  devStates[sid] = localState;
  sum[sid] = tmp;

  // do reduction on GPU
  __reduction_on_gpu(sum, sid, result[blockIdx.x]);
}

__global__ void curand_setup_kernel(hiprandStateMRG32k3a *devStates,
                                    unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);
}

//-----------------------------------------------------------------------------
//  Curand Philox
//-----------------------------------------------------------------------------
__global__ void KernelCurandPhilox(hiprandStatePhilox4_32_10_t *devStates,
                                   double *result, int nsample) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  unsigned int sid = threadIdx.x;

  hiprandStatePhilox4_32_10_t localState = devStates[tid];

  __shared__ double sum[THREADS_PER_BLOCK];
  double tmp = 0;

  while (tid < nsample) {
    tmp += hiprand_uniform_double(&localState);
    tid += blockDim.x * gridDim.x;
  }
  devStates[sid] = localState;
  sum[sid] = tmp;

  // do reduction on GPU
  __reduction_on_gpu(sum, sid, result[blockIdx.x]);
}

__global__ void curand_setup_kernel(hiprandStatePhilox4_32_10_t *devStates,
                                    unsigned long seed) {
  unsigned int tid = threadIdx.x + blockIdx.x * blockDim.x;
  hiprand_init(seed, tid, 0, &devStates[tid]);
}

} // end namespace cuda

// Cuda wrapper

void CudaMRG32k3a(vecRng::MRG32k3a<vecRng::ScalarBackend>::State_t *devStates,
                  double *result, int nsample, int blocksPerGrid,
                  int threadsPerBlock) {
  KernelMRG32k3a<<<blocksPerGrid, threadsPerBlock>>>(devStates, result,
                                                     nsample);
}

void CudaThreefry(vecRng::Threefry<vecRng::ScalarBackend>::State_t *devStates,
                  double *result, int nsample, int blocksPerGrid,
                  int threadsPerBlock) {
  KernelThreefry<<<blocksPerGrid, threadsPerBlock>>>(devStates, result,
                                                     nsample);
}

void CudaPhilox(vecRng::Philox<vecRng::ScalarBackend>::State_t *devStates,
                double *result, int nsample, int blocksPerGrid,
                int threadsPerBlock) {
  KernelPhilox<<<blocksPerGrid, threadsPerBlock>>>(devStates, result, nsample);
}

//-----------------------------------------------------------------------------
//  cuda wrapper for Curand MRG32k3a Kernel
//-----------------------------------------------------------------------------
void CurandMRG32k3a(hiprandStateMRG32k3a *devStates, double *result, int nsample,
                    int blocksPerGrid, int threadsPerBlock) {
  int kstatus = 0;

  KernelCurandMRG32k3a<<<blocksPerGrid, threadsPerBlock>>>(devStates, result,
                                                           nsample);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "MRG32k3a_gpu status = " << kstatus << "\n";
}

void curand_setup_gpu(hiprandStateMRG32k3a *devStates, unsigned long seed,
                      int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<<blocksPerGrid, threadsPerBlock>>>(devStates, seed);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "MRG32k3a: cuda_setup_kernel status = " << kstatus << "\n";
}

//-----------------------------------------------------------------------------
//  cuda wrapper for Curand Philox4_32_10 Kernel
//-----------------------------------------------------------------------------
void CurandPhilox(hiprandStatePhilox4_32_10_t *devStates, double *result,
                  int nsample, int blocksPerGrid, int threadsPerBlock) {
  int kstatus = 0;

  KernelCurandPhilox<<<blocksPerGrid, threadsPerBlock>>>(devStates, result,
                                                         nsample);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "CurandPhilox status = " << kstatus << "\n";
}

void curand_setup_gpu(hiprandStatePhilox4_32_10_t *devStates, unsigned long seed,
                      int NBLOCKS, int NTHREADS) {

  int kstatus = 0;

  int threadsPerBlock = NTHREADS;
  int blocksPerGrid = NBLOCKS;

  curand_setup_kernel<<<blocksPerGrid, threadsPerBlock>>>(devStates, seed);

  kstatus = hipDeviceSynchronize();
  if (kstatus)
    std::cout << "Philox: cuda_setup_kernel status = " << kstatus << "\n";
}

} // end namespace vecrng
